#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  projection.cu
 *
 *    Description:  Este arquivo contem a implementacao do operador projection em CUDA/GPU
 *
 *        Version:  1.0
 *        Created:  06/02/16 08:55:22
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Victor Silva
 *
 * =====================================================================================
 */
#include "projection.h"

__global__ void projectColumns ( float		*tupla,
								 float		*novaTupla,
								 int		*colunas,
								 int 		size )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		novaTupla[i] = tupla[ colunas[i] ];
	}
}

Relation project( Relation inputRelation, vector<int> columns ) {
	Relation outputRelation;
	vector<float> h_outputRelationHeaders;

	for(int i = 0; i < columns.size(); i++){
		int colIndex = columns[i];
		outputRelation.addColumn( inputRelation.getColumnName(colIndex));
	}

	thrust::device_vector<int> d_columns(columns);
	int *ptr_columns = thrust::raw_pointer_cast( d_columns.data() );

	for(int i = 0; i < inputRelation.size(); i++){
		thrust::device_vector<float> d_tupla( inputRelation.getTupla(i) );
		thrust::device_vector<float> d_newTupla(columns.size(), 0);
		vector<float> h_newTupla(columns.size(), 0);

		float *ptr_tupla = thrust::raw_pointer_cast( d_tupla.data() );
		float *ptr_newTupla = thrust::raw_pointer_cast( d_newTupla.data() );

		projectColumns<<< 1, columns.size() >>>( ptr_tupla, ptr_newTupla, ptr_columns, columns.size() );

		thrust::copy( d_newTupla.begin(), d_newTupla.end(), h_newTupla.begin() );


		outputRelation.addTupla( h_newTupla );
	}

	return outputRelation;

}