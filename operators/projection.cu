#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  projection.cu
 *
 *    Description:  Este arquivo contem a implementacao do operador projection em CUDA/GPU
 *
 *        Version:  1.0
 *        Created:  06/02/16 08:55:22
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Victor Silva
 *
 * =====================================================================================
 */
#include "projection.h"

__global__ void copyHeaders (	string	inputRelationHeaders, 
								int 	*columns,
							  	string 	*newHeader,
							  	int 	size )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size ) {
		newHeader[index] = inputRelationHeaders[columns[index]];
	}
}

__global__ void projectColumns ( string		*tupla,
								 int		tSize,
								 int		*columns,
								 int		cSize,
								 string 	*novaTupla)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < tSize) {
		for (int i = 0; i < cSize; i++) {
			novaTupla[i] = tupla[columns[i]];
		}
	}
}

Relation project( Relation inputRelation, vector<int> columns ) {
	Relation outputRelation;
	vector<string> h_outputRelationHeaders;

	const unsigned int numObjs = inputRelation.size();
	const unsigned int numThreadsPerClusterBlock = 128;
    const unsigned int numClusterBlocks =
        (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;
    const unsigned int clusterBlockSharedDataSize =
        numThreadsPerClusterBlock * sizeof(unsigned char);

	thrust::device_vector<string> d_inputRelationHeaders(inputRelation.getHeaders());
	thrust::device_vector<int> d_columns(columns);
	thrust::device_vector<string> d_newHeader(columns.size());

	string *ptr_inputRelationHeaders = thrust::raw_pointer_cast( d_inputRelationHeaders.data() );
	int *ptr_columns = thrust::raw_pointer_cast( d_columns.data() );
	string *ptr_newHeader = thrust::raw_pointer_cast( d_newHeader.data() );

	copyHeaders<<< 1, columns.size() >>>( ptr_inputRelationHeaders, ptr_columns, ptr_newHeader, columns.size() );

	thrust::copy( d_newHeader.begin(), d_newHeader.end(), h_outputRelationHeaders.begin() );
	outputRelation.setHeaders( h_outputRelationHeaders );

	for(int i = 0; i < inputRelation.size(); i++){
		thrust::device_vector<string> d_tupla( inputRelation.getTupla(i) );
		thrust::device_vector<string> d_newTupla;
		vector<string> h_newTupla;

		string *ptr_tupla = thrust::raw_pointer_cast( d_tupla.data() );
		string *ptr_newTupla = thrust::raw_pointer_cast( d_newTupla.data() );

		projectColumns
			<<< numClusterBlocks, numThreadsPerClusterBlock >>> 
			( ptr_tupla, inputRelation.size(), ptr_columns, columns.size(), ptr_newTupla );

		thrust::copy( d_newTupla.begin(), d_newTupla.end(), h_newTupla.begin() );

		outputRelation.addTupla( h_newTupla );
	}

	return outputRelation;


}