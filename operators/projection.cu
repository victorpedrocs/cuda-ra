#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  projection.cu
 *
 *    Description:  Este arquivo contem a implementacao do operador projection em CUDA/GPU
 *
 *        Version:  1.0
 *        Created:  06/02/16 08:55:22
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Victor Silva
 *
 * =====================================================================================
 */
#include "projection.h"

__global__ void projectColumns ( float		*tupla,
								 float		*novaTupla,
								 int		*colunas,
								 int 		size )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		novaTupla[i] = tupla[ colunas[i] ];
	}
}

Relation project( Relation inputRelation, vector<int> columns ) {
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float totalMillis = 0;

	Relation outputRelation;
	vector<float> h_outputRelationHeaders;


	for(int i = 0; i < columns.size(); i++){
		int colIndex = columns[i];
		outputRelation.addColumn( inputRelation.getColumnName(colIndex));
	}

	thrust::device_vector<int> d_columns(columns);
	int *ptr_columns = thrust::raw_pointer_cast( d_columns.data() );

	for(int i = 0; i < inputRelation.size(); i++){
		thrust::device_vector<float> d_tupla( inputRelation.getTupla(i) );
		thrust::device_vector<float> d_newTupla(columns.size(), 0);
		vector<float> h_newTupla(columns.size(), 0);

		float *ptr_tupla = thrust::raw_pointer_cast( d_tupla.data() );
		float *ptr_newTupla = thrust::raw_pointer_cast( d_newTupla.data() );

		checkCuda(hipEventRecord(start));
		projectColumns<<< 1, columns.size() >>>( ptr_tupla, ptr_newTupla, ptr_columns, columns.size() );
		checkCuda(hipEventRecord(stop));

		thrust::copy( d_newTupla.begin(), d_newTupla.end(), h_newTupla.begin() );


		outputRelation.addTupla( h_newTupla );

		checkCuda(hipEventSynchronize(stop));
		float milliseconds = 0;
		checkCuda(hipEventElapsedTime(&milliseconds, start, stop));
		totalMillis += milliseconds;
	}

	checkCuda( hipEventDestroy( start ) );
	checkCuda( hipEventDestroy( stop ) );
	float seconds = totalMillis/1000;
	cout << "GPU Time: " << seconds << endl;

	return outputRelation;

}