#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  projection.cu
 *
 *    Description:  Este arquivo contem a implementacao do operador projection em CUDA/GPU
 *
 *        Version:  1.0
 *        Created:  06/02/16 08:55:22
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Victor Silva
 *
 * =====================================================================================
 */
#include "projection.h"

__global__ void projectColumns ( float		*oldRelation,
								 float		*newRelation,
								 int		*columns,
								 int 		sizeAllTuplas,
								 int 		columnsSize,
								 int 		tuplaSize )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < sizeAllTuplas ) {
		int iTupla = i/tuplaSize;
		for (int it = 0; it < columnsSize; it++) {
			newRelation[it + iTupla] = oldRelation[ columns[it] + iTupla ];
		}
	}
}

Relation project( Relation inputRelation, vector<int> columns ) {
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float totalMillis = 0;

	Relation outputRelation;
	vector<float> h_outputRelationHeaders;

	// Copia os headers para a nova relação
	for(int i = 0; i < columns.size(); i++){
		int colIndex = columns[i];
		outputRelation.addColumn( inputRelation.getColumnName(colIndex));
	}
	
	const unsigned int sizeAllTuplas = inputRelation.size() * inputRelation.getHeaders().size();
	const unsigned int relationSize = inputRelation.size();
	const unsigned int tuplaSize = inputRelation.getHeaders().size();

	thrust::host_vector<float> h_oldRelation(sizeAllTuplas, 0);
	thrust::device_vector<int> d_columns(columns);	
	thrust::device_vector<float> d_oldRelation( sizeAllTuplas );
	thrust::device_vector<float> d_newRelation( inputRelation.size() * columns.size(), 0 );
	
	for (int i = 0; i < sizeAllTuplas; i++) {
		int tuplaIndex = i/tuplaSize;
		int col = i%tuplaSize;
		h_oldRelation[i] = inputRelation.getTupla(tuplaIndex)[col];
	}
	thrust::copy( h_oldRelation.begin(), h_oldRelation.end(), d_oldRelation.begin() );

	int *ptr_columns = thrust::raw_pointer_cast( d_columns.data() );
	float *ptr_oldRelation = thrust::raw_pointer_cast( d_oldRelation.data() );
	float *ptr_newRelation = thrust::raw_pointer_cast( d_newRelation.data() );

	const unsigned int numThreadsPerClusterBlock = 256;
	const unsigned int numClusterBlocks = (sizeAllTuplas + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;
	
	cout << "Chegou no Kernel" << endl;
	checkCuda(hipEventRecord(start));
	projectColumns
		<<< numClusterBlocks, numThreadsPerClusterBlock >>>
		( ptr_oldRelation, ptr_newRelation, ptr_columns, sizeAllTuplas, columns.size(), tuplaSize );
	checkCuda(hipEventRecord(stop));
	checkCuda(hipEventSynchronize(stop));

	thrust::host_vector<float> h_newRelation( d_newRelation );
	vector< vector<float> > newTuplas(relationSize);
	for (int i = 0; i < sizeAllTuplas; i++)
	{
		int tuplaIndex = i/tuplaSize;
		newTuplas[tuplaIndex].push_back(h_newRelation[i]);
	}

	outputRelation.setTuplas(newTuplas);

	checkCuda(hipEventElapsedTime(&totalMillis, start, stop));
	checkCuda( hipEventDestroy( start ) );
	checkCuda( hipEventDestroy( stop ) );
	double seconds = totalMillis/1000;
	printf("GPU Time =\t%f\n", seconds);

	return outputRelation;

}