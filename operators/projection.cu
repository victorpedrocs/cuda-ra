#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  projection.cu
 *
 *    Description:  Este arquivo contem a implementacao do operador projection em CUDA/GPU
 *
 *        Version:  1.0
 *        Created:  06/02/16 08:55:22
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Victor Silva
 *
 * =====================================================================================
 */
#include "projection.h"

__global__ void makeProjection (){}
